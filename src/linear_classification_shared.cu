#include "hip/hip_runtime.h"
#include "linear_classification.cuh"

using namespace std;

// Number of thread per block
#define THREADS_PER_BLOCK 1024
/* Constants for housing data set */
#define data_columns (9)
#define above_threshold (265000.0)

/////////////////////////////////////////////////////////
// Number of rows in arrays to print for debugging
/////////////////////////////////////////////////////////
#define print_rows (10)

/* Main function */
extern "C"
{
    float linear_classification(unsigned nb_iter, unsigned batch_size_in, float learning_rate, float rate_decay)
    {
        /* Parameters for the data set */
        unsigned N_train = 12000; // 12000; // points for training (Google: 12000)
        unsigned N_test = 5000;   // 5000; // points for validation (Google: 5000)
        unsigned N = N_train;
        unsigned Nall = N_train + N_test;

        /* Clamp batch size between 1 and N */
        unsigned batch_size = min(N, max(1u, batch_size_in));

        /* Hyperarameters for Stochastic Gradient Descent */
        unsigned periods = nb_iter; // reporting period
        bool verbose = false;       // Show logs

        /* Reading the data set */
        fmatrix alldata = fmatrix_create_on_host(Nall, data_columns);
        read_csv("sample_data/california_housing_train.csv", alldata.data, Nall, data_columns);

        unsigned D = data_columns - 1 + 1; // remove output column, add column with const. 1.0
        unsigned M = 2;                    // number of labels (one-hot encoding)
        fmatrix Xall = fmatrix_create_on_host((int)D, (int)Nall);
        fmatrix Yall = fmatrix_create_on_host((int)M, (int)Nall);
        get_inputs_and_labels(alldata.data, &Xall.data, &Yall.data, Nall, data_columns, D, M);

        // Logfile
        FILE *fp = fopen("log.txt", "w");

        /* Memory Allocation and Initialization */
        fmatrix h_X = fmatrix_subcolumns(Xall, 0, (int)N);
        fmatrix h_Y = fmatrix_subcolumns(Yall, 0, (int)N);
        fmatrix h_Xtest = fmatrix_subcolumns(Xall, (int)N, (int)Nall);
        fmatrix h_Ytest = fmatrix_subcolumns(Yall, (int)N, (int)Nall);
        fmatrix h_W = fmatrix_create_on_host((int)D, (int)M);
        fmatrix h_J = fmatrix_create_on_host(1, 1);

        xavier_weight_init(1.0, h_W);

        /* Copy data to device */
        fmatrix d_X_tot = fmatrix_copy_to_device(h_X);
        fmatrix d_Y_tot = fmatrix_copy_to_device(h_Y);
        fmatrix d_Xtest = fmatrix_copy_to_device(h_Xtest);
        fmatrix d_Ytest = fmatrix_copy_to_device(h_Ytest);
        fmatrix d_W = fmatrix_copy_to_device(h_W);
        fmatrix d_J = fmatrix_copy_to_device(h_J);

        /* Normalize */
        fmatrix d_Mu = compute_mean(d_X_tot);
        fmatrix d_Std = compute_std(d_X_tot, d_Mu);

        parametered_normalize(d_X_tot, d_Mu, d_Std);
        parametered_normalize(d_Xtest, d_Mu, d_Std);
        gpuErrchk(hipPeekAtLastError());

        /* Create auxiliary matrices on device */
        fmatrix d_Z = fmatrix_create_on_device((int)M, (int)batch_size);
        fmatrix d_P = fmatrix_create_on_device((int)M, (int)batch_size);
        fmatrix d_G = fmatrix_create_on_device((int)D, (int)M);
        fmatrix d_Ztest = fmatrix_create_on_device((int)M, d_Xtest.cols);

        /////////////////////////////////////////////////////////
        // Batch Gradient Descent
        /////////////////////////////////////////////////////////
        // fmatrix_device_print(d_X);
        // fmatrix_device_print(d_W);

        /* Create Handle */
        hipblasHandle_t handle;
        hipblasStatus_t stat = hipblasCreate(&handle);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            printf("CUBLAS initialisation failed\n");
        }

        /* Evaluate the starting accuracy */
        float accuracy = 0;
        accuracy = evaluate_accuracy(handle, d_W, d_Xtest, d_Ytest, d_Ztest, verbose);

        float alpha = 1.0f;
        float beta = 0.0f;

        clock_t t_start_total, t_end;
        t_start_total = clock();

        for (unsigned i = 0; i < nb_iter; ++i)
        {
            /* Reset the values for a new epoch */
            unsigned batch_pointer = 0;
            float J = 0;
            unsigned nb_col = batch_size;
            d_P.cols = (int)batch_size;
            d_Z.cols = (int)batch_size;

            /* Loop on mini-batches */
            while (batch_pointer < N)
            {
                /* If the batch is smaller */
                if (batch_pointer + batch_size > N)
                {
                    nb_col = N - batch_pointer;
                    d_P.cols = (int)nb_col;
                    d_Z.cols = (int)nb_col;
                }
                fmatrix d_X = fmatrix_subcolumns(d_X_tot, batch_pointer, batch_pointer + nb_col);
                fmatrix d_Y = fmatrix_subcolumns(d_Y_tot, batch_pointer, batch_pointer + nb_col);

                /* compute Z = W^T X */
                hipblasStatus_t multstat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, d_W.cols, d_X.cols, d_W.rows, &alpha, d_W.data, d_W.rows, d_X.data, d_X.rows, &beta, d_Z.data, d_Z.rows);
                gpuErrchk(hipPeekAtLastError());

                if (multstat != HIPBLAS_STATUS_SUCCESS)
                {
                    printf("CUBLAS matrix multiplication failed 3 %d\n", multstat);
                }

                /* compute softmax per column of Z and store in P */
                d_P = softmax_col(d_Z);
                gpuErrchk(hipPeekAtLastError());

                // Q := P-Y
                fmatrix d_Q = fmatrix_add(d_P, -1.0f, d_Y);
                gpuErrchk(hipPeekAtLastError());

                // compute gradient G = XQ^T
                multstat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, d_X.rows, d_Q.rows, d_X.cols, &alpha, d_X.data, d_X.rows, d_Q.data, d_Q.rows, &beta, d_G.data, d_G.rows);
                gpuErrchk(hipPeekAtLastError());

                if (multstat != HIPBLAS_STATUS_SUCCESS)
                {
                    printf("CUBLAS matrix multiplication failed 4\n");
                }

                // update weights W = W - learning_rate*G
                d_W = fmatrix_add(d_W, -learning_rate, d_G);
                gpuErrchk(hipPeekAtLastError());

                /* Compute J for reporting */
                J = evaluate_logloss(handle, d_P, d_Y, verbose);

                /* Increase the pointer */
                batch_pointer += batch_size;
            }

            /* Log accuracy and loss */
            if (i % (nb_iter / periods) == 0)
            {
                float accuracy = evaluate_accuracy(handle, d_W, d_Xtest, d_Ytest, d_Ztest, verbose);
                fprintf(fp, "%f,%f\n", J, accuracy);
            }

            /* reduce the learning rate */
            learning_rate *= rate_decay;
        }
        t_end = clock();
        float duration = ((float)(t_end - t_start_total)) / CLOCKS_PER_SEC;
        /* Evaluate the accuracy */
        accuracy = evaluate_accuracy(handle, d_W, d_Xtest, d_Ytest, d_Ztest, verbose);

        /* Memory clean up */
        fmatrix_free_on_host(&h_W);
        fmatrix_free_on_host(&Xall);
        fmatrix_free_on_host(&Yall);

        fmatrix_free_on_device(&d_X_tot);
        fmatrix_free_on_device(&d_Y_tot);
        fmatrix_free_on_device(&d_Xtest);
        fmatrix_free_on_device(&d_Ytest);
        fmatrix_free_on_device(&d_W);
        fmatrix_free_on_device(&d_Z);
        fmatrix_free_on_device(&d_J);
        hipblasDestroy(handle);

        // Close log file
        fclose(fp);
        return accuracy;
    }
}