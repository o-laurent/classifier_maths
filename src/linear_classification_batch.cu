#include "hip/hip_runtime.h"
#include "linear_classification.cuh"

using namespace std;

// Number of thread per block
#define THREADS_PER_BLOCK 1024
/* Constants for housing data set */
#define data_columns (9)
#define above_threshold (265000.0)

/////////////////////////////////////////////////////////
// Number of rows in arrays to print for debugging
/////////////////////////////////////////////////////////
#define print_rows (10)

/////////////////////////////////////////////////////////
// Main program
/////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    /* Parameters for the data set */
    unsigned N_train = 12000; // 12000; // points for training (Google: 12000)
    unsigned N_test = 5000;   // 5000; // points for validation (Google: 5000)
    unsigned N = N_train;
    unsigned Nall = N_train + N_test;

    /* Hyperarameters for Stochastic Gradient Descent */
    unsigned nb_iter = 10;        // default: 10;
    unsigned periods = nb_iter;   // reporting period
    unsigned batch_size = 8;      // default: N;
    float learning_rate = 0.057f; // default: 1e-7
    float rate_decay = 0.59f;
    bool verbose = false; // Show logs

    /* Reading the data set */
    fmatrix alldata = fmatrix_create_on_host(Nall, data_columns);
    read_csv("sample_data/california_housing_train.csv", alldata.data, Nall, data_columns);

    unsigned D = data_columns - 1 + 1; // remove output column, add column with const. 1.0
    unsigned M = 2;                    // number of labels (one-hot encoding)
    fmatrix Xall = fmatrix_create_on_host((int)D, (int)Nall);
    fmatrix Yall = fmatrix_create_on_host((int)M, (int)Nall);
    get_inputs_and_labels(alldata.data, &Xall.data, &Yall.data, Nall, data_columns, D, M);

    // Logfile
    FILE *fp = fopen("log.txt", "w");

    /* Memory Allocation and Initialization */
    fmatrix h_X = fmatrix_subcolumns(Xall, 0, (int)N);
    fmatrix h_Y = fmatrix_subcolumns(Yall, 0, (int)N);
    fmatrix h_Xtest = fmatrix_subcolumns(Xall, (int)N, (int)Nall);
    fmatrix h_Ytest = fmatrix_subcolumns(Yall, (int)N, (int)Nall);
    fmatrix h_W = fmatrix_create_on_host((int)D, (int)M);

    xavier_weight_init(1.0, h_W);

    /* Copy data to device */
    fmatrix d_X_tot = fmatrix_copy_to_device(h_X);
    fmatrix d_Y_tot = fmatrix_copy_to_device(h_Y);
    fmatrix d_Xtest = fmatrix_copy_to_device(h_Xtest);
    fmatrix d_Ytest = fmatrix_copy_to_device(h_Ytest);
    fmatrix d_W = fmatrix_copy_to_device(h_W);

    /* Normalize */
    fmatrix d_Mu = compute_mean(d_X_tot);
    fmatrix d_Std = compute_std(d_X_tot, d_Mu);

    if (verbose)
    {
        printf("d_W default:\n");
        fmatrix_device_print(d_W);

        printf("d_X:\n");
        fmatrix_device_print(d_X_tot);

        printf("d_Mu:\n");
        fmatrix_device_print(d_Mu);

        printf("d_Std:\n");
        fmatrix_device_print(d_Std);
    }

    parametered_normalize(d_X_tot, d_Mu, d_Std);
    parametered_normalize(d_Xtest, d_Mu, d_Std);

    /* Create auxiliary matrices on device */
    fmatrix d_Z = fmatrix_create_on_device((int)M, (int)batch_size);
    fmatrix d_P = fmatrix_create_on_device((int)M, (int)batch_size);
    fmatrix d_G = fmatrix_create_on_device((int)D, (int)M);
    fmatrix d_Ztest = fmatrix_create_on_device((int)M, d_Xtest.cols);

    /* Create Handle */
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS initialisation failed\n");
    }

    /* Evaluate the starting accuracy */
    float accuracy = 0;
    accuracy = evaluate_accuracy(handle, d_W, d_Xtest, d_Ytest, d_Ztest, verbose);
    printf("Initial accuracy: %f\n", accuracy);

    float alpha = 1.0f;
    float beta = 0.0f;

    clock_t t_start_total, t_end;
    t_start_total = clock();

    for (unsigned i = 0; i < nb_iter; ++i)
    {
        unsigned batch_pointer = 0;
        float J = 0;
        unsigned nb_col = batch_size;
        d_P.cols = (int)batch_size;
        d_Z.cols = (int)batch_size;

        /* Loop on mini-batches */
        while (batch_pointer < N)
        {
            /* If the batch is smaller */
            if (batch_pointer + batch_size > N)
            {
                nb_col = N - batch_pointer;
                d_P.cols = (int)nb_col;
                d_Z.cols = (int)nb_col;
            }
            fmatrix d_X = fmatrix_subcolumns(d_X_tot, batch_pointer, batch_pointer + nb_col);
            fmatrix d_Y = fmatrix_subcolumns(d_Y_tot, batch_pointer, batch_pointer + nb_col);

            /* compute Z = W^T X */
            hipblasStatus_t multstat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, d_W.cols, d_X.cols, d_W.rows, &alpha, d_W.data, d_W.rows, d_X.data, d_X.rows, &beta, d_Z.data, d_Z.rows);
            if (verbose)
            {
                printf("d_W:");
                fmatrix_device_print(d_W);

                printf("d_X:");
                fmatrix_device_print(d_X);

                printf("dw rows %d, dw cols %d, dX rows %d, dX cols %d, dZ rows %d, dZ cols %d\n", d_W.rows, d_W.cols, d_X.rows, d_X.cols, d_Z.rows, d_Z.cols);
                printf("m %d, n %d, k %d\n", d_W.cols, d_X.cols, d_W.rows);

                printf("d_Z:");
                fmatrix_device_print(d_Z);
            }
            if (multstat != HIPBLAS_STATUS_SUCCESS)
            {
                printf("CUBLAS matrix multiplication failed 3 %d\n", multstat);
                gpuErrchk(hipPeekAtLastError());
            }

            if (verbose)
            {
                printf("d_P.rows: %d d_P.cols: %d, d_Y.rows: %d, d_Y.cols: %d\n", d_P.rows, d_P.cols, d_Y.rows, d_Y.cols);
            }
            // compute softmax per column of Z and store in P
            d_P = softmax_col(d_Z);
            gpuErrchk(hipPeekAtLastError());

            // Q := P-Y
            fmatrix d_Q = fmatrix_add(d_P, -1.0f, d_Y);
            gpuErrchk(hipPeekAtLastError());

            if (verbose)
            {
                printf("d_P:");
                fmatrix_device_print(d_P);

                printf("d_Y:");
                fmatrix_device_print(d_Y);

                printf("d_Q:");
                fmatrix_device_print(d_Q);
            }

            /* compute gradient G = XQ^T */
            multstat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, d_X.rows, d_Q.rows, d_X.cols, &alpha, d_X.data, d_X.rows, d_Q.data, d_Q.rows, &beta, d_G.data, d_G.rows);
            gpuErrchk(hipPeekAtLastError());
            if (verbose)
            {
                printf("d_X:");
                fmatrix_device_print(d_X);

                printf("d_Q:");
                fmatrix_device_print(d_Q);

                printf("d_G:");
                fmatrix_device_print(d_G);

                printf("dw rows %d, dw cols %d, dX rows %d, dX cols %d, dG rows %d, dG cols %d\n", d_X.rows, d_X.cols, d_Q.rows, d_Q.cols, d_G.rows, d_G.cols);
                printf("m %d, n %d, k %d\n", d_X.rows, d_Q.rows, d_X.cols);
            }
            if (multstat != HIPBLAS_STATUS_SUCCESS)
            {
                printf("CUBLAS matrix multiplication failed 4\n");
                gpuErrchk(hipPeekAtLastError());
            }

            /* update weights W = W - learning_rate*G */
            d_W = fmatrix_add(d_W, -learning_rate, d_G);
            gpuErrchk(hipPeekAtLastError());

            if (verbose)
            {
                fmatrix_device_print(d_W);
            }

            /* Compute J for reporting */
            J = evaluate_logloss(handle, d_P, d_Y, verbose);

            /* Increase pointer */
            batch_pointer += batch_size;
        }
        /* Log accuracy and loss */
        if (i % (nb_iter / periods) == 0)
        {
            float accuracy = evaluate_accuracy(handle, d_W, d_Xtest, d_Ytest, d_Ztest, verbose);
            printf("iter: %u, logloss: %f, accuracy: %f, lr: %f\n", i, J, accuracy, learning_rate);
            fprintf(fp, "%f,%f\n", J, accuracy);
        }

        /* reduce the learning rate */
        learning_rate *= rate_decay;
    }
    t_end = clock();
    float duration = ((float)(t_end - t_start_total)) / CLOCKS_PER_SEC;
    printf("Duration (s): %f\n", duration);
    /* Evaluate the accuracy */
    accuracy = evaluate_accuracy(handle, d_W, d_Xtest, d_Ytest, d_Ztest, verbose);
    printf("Final accuracy on test set: %f\n", accuracy);

    printf("Final weights: \n");
    fmatrix_device_print(d_W);

    /* Memory clean up */
    fmatrix_free_on_host(&h_W);
    fmatrix_free_on_host(&Xall);
    fmatrix_free_on_host(&Yall);

    fmatrix_free_on_device(&d_X_tot);
    fmatrix_free_on_device(&d_Y_tot);
    fmatrix_free_on_device(&d_Xtest);
    fmatrix_free_on_device(&d_Ytest);
    fmatrix_free_on_device(&d_W);
    fmatrix_free_on_device(&d_Z);
    fmatrix_free_on_device(&d_Mu);
    fmatrix_free_on_device(&d_Std);

    hipblasDestroy(handle);

    // Close log file
    fclose(fp);
}